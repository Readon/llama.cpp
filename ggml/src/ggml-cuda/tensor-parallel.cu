#include "hip/hip_runtime.h"
#include "tensor-parallel.cuh"
#include "nccl.cuh"
#include <algorithm>
#include <cstring>

// Global tensor parallelism context
std::unique_ptr<ggml_backend_cuda_tp_context> g_cuda_tp_ctx = nullptr;

// Tensor name patterns for different TP strategies
namespace ggml_tp_patterns {
    // Column-wise split patterns (output projections, feed-forward layers)
    const char* column_split_patterns[] = {
        ".attn_output.weight",
        ".ffn_down.weight", 
        ".ffn_gate.weight",
        ".ffn_up.weight",
        ".output.weight",
        nullptr
    };
    
    // Row-wise split patterns (input projections)
    const char* row_split_patterns[] = {
        ".attn_q.weight",
        ".attn_k.weight", 
        ".attn_v.weight",
        ".attn_qkv.weight",
        nullptr
    };
    
    // Replicate patterns (embeddings, layer norms, biases)
    const char* replicate_patterns[] = {
        ".tok_embd.weight",
        ".norm.weight",
        ".norm.bias",
        ".attn_norm.weight",
        ".ffn_norm.weight",
        ".output_norm.weight",
        nullptr
    };
    
    bool matches_pattern(const std::string& tensor_name, const char* patterns[]) {
        for (int i = 0; patterns[i] != nullptr; i++) {
            if (tensor_name.find(patterns[i]) != std::string::npos) {
                return true;
            }
        }
        return false;
    }
}

ggml_tp_strategy ggml_get_tensor_parallel_strategy(const std::string& tensor_name,
                                                   const struct ggml_tensor* tensor,
                                                   const ggml_tp_config& tp_config) {
    if (!tp_config.enabled) {
        return GGML_TP_STRATEGY_REPLICATE;
    }
    
    // Check for explicit patterns first
    if (ggml_tp_patterns::matches_pattern(tensor_name, ggml_tp_patterns::column_split_patterns)) {
        return GGML_TP_STRATEGY_COLUMN;
    }
    
    if (ggml_tp_patterns::matches_pattern(tensor_name, ggml_tp_patterns::row_split_patterns)) {
        return GGML_TP_STRATEGY_ROW;
    }
    
    if (ggml_tp_patterns::matches_pattern(tensor_name, ggml_tp_patterns::replicate_patterns)) {
        return GGML_TP_STRATEGY_REPLICATE;
    }
    
    // Auto-determine strategy based on tensor properties
    if (tensor->ne[0] % tp_config.tp_size == 0 && tensor->ne[0] >= tp_config.tp_size) {
        // Can split along first dimension
        return GGML_TP_STRATEGY_ROW;
    } else if (tensor->ne[1] % tp_config.tp_size == 0 && tensor->ne[1] >= tp_config.tp_size) {
        // Can split along second dimension  
        return GGML_TP_STRATEGY_COLUMN;
    }
    
    // Default to replication
    return GGML_TP_STRATEGY_REPLICATE;
}

bool ggml_tensor_supports_tp(const std::string& tensor_name, const struct ggml_tensor* tensor) {
    // Only support tensor parallelism for 2D weight matrices
    if (ggml_n_dims(tensor) != 2) {
        return false;
    }

    // Skip very small tensors
    if (ggml_nelements(tensor) < 1024) {
        return false;
    }

    // Check if it's a weight tensor (not bias or other parameters)
    return tensor_name.find(".weight") != std::string::npos;
}

ggml_tp_split_info ggml_calculate_tp_split(const struct ggml_tensor* tensor,
                                          ggml_tp_strategy strategy,
                                          const ggml_tp_config& tp_config) {
    ggml_tp_split_info info = {};
    info.split_dim = -1;
    info.split_size = 0;
    info.split_offset = 0;
    info.needs_all_reduce = false;
    info.needs_all_gather = false;
    
    if (!tp_config.enabled || strategy == GGML_TP_STRATEGY_REPLICATE) {
        return info;
    }
    
    switch (strategy) {
        case GGML_TP_STRATEGY_COLUMN:
            if (tensor->ne[1] % tp_config.tp_size == 0) {
                info.split_dim = 1;
                info.split_size = tensor->ne[1] / tp_config.tp_size;
                info.split_offset = tp_config.tp_rank * info.split_size;
                info.needs_all_reduce = true;
            }
            break;
            
        case GGML_TP_STRATEGY_ROW:
            if (tensor->ne[0] % tp_config.tp_size == 0) {
                info.split_dim = 0;
                info.split_size = tensor->ne[0] / tp_config.tp_size;
                info.split_offset = tp_config.tp_rank * info.split_size;
                info.needs_all_gather = true;
            }
            break;
            
        default:
            break;
    }
    
    return info;
}

bool ggml_apply_tensor_parallel_split(struct ggml_tensor* tensor,
                                     const ggml_tp_config& tp_config,
                                     ggml_tp_strategy strategy) {
    if (!tp_config.enabled || strategy == GGML_TP_STRATEGY_REPLICATE) {
        return true;
    }
    
    ggml_tp_split_info split_info = ggml_calculate_tp_split(tensor, strategy, tp_config);
    
    if (split_info.split_dim == -1) {
        return false; // Cannot split this tensor
    }
    
    // Modify tensor dimensions to reflect the split
    if (split_info.split_dim == 0) {
        tensor->ne[0] = split_info.split_size;
    } else if (split_info.split_dim == 1) {
        tensor->ne[1] = split_info.split_size;
    }
    
    // Recalculate strides
    tensor->nb[0] = ggml_type_size(tensor->type);
    for (int i = 1; i < GGML_MAX_DIMS; i++) {
        tensor->nb[i] = tensor->nb[i-1] * tensor->ne[i-1];
    }
    
    return true;
}

namespace ggml_tp_utils {
    int64_t get_split_elements(int64_t total_elements, int tp_size, int tp_rank) {
        int64_t base_size = total_elements / tp_size;
        int64_t remainder = total_elements % tp_size;
        
        if (tp_rank < remainder) {
            return base_size + 1;
        } else {
            return base_size;
        }
    }
    
    int64_t get_split_offset(int64_t total_elements, int tp_size, int tp_rank) {
        int64_t base_size = total_elements / tp_size;
        int64_t remainder = total_elements % tp_size;
        
        int64_t offset = tp_rank * base_size;
        if (tp_rank < remainder) {
            offset += tp_rank;
        } else {
            offset += remainder;
        }
        
        return offset;
    }
    
    bool check_tp_compatibility(const struct ggml_tensor* tensor, int tp_size, int split_dim) {
        if (split_dim < 0 || split_dim >= ggml_n_dims(tensor)) {
            return false;
        }

        return tensor->ne[split_dim] % tp_size == 0;
    }
}

ggml_backend_cuda_tp_context::ggml_backend_cuda_tp_context(int tp_size, const std::vector<int>& devices)
    : config(tp_size, 0), device_ids(devices), nccl_initialized(false) {
}

ggml_backend_cuda_tp_context::~ggml_backend_cuda_tp_context() {
    cleanup();
}

bool ggml_backend_cuda_tp_context::init() {
    if (config.tp_size <= 1) {
        return true;
    }

    // Try to initialize NCCL for collective operations
    nccl_initialized = ggml_cuda_nccl_init(device_ids);

    if (!nccl_initialized) {
        GGML_LOG_INFO("NCCL not available, using basic tensor parallelism mode\n");
        GGML_LOG_INFO("Note: Install NCCL for optimized collective operations\n");
        // Continue without NCCL - basic tensor parallelism can still work
    } else {
        GGML_LOG_INFO("NCCL initialized for optimized tensor parallelism\n");
    }

    GGML_LOG_INFO("Tensor parallelism initialized: %d-way TP using GPUs ", config.tp_size);
    for (size_t i = 0; i < device_ids.size(); i++) {
        GGML_LOG_INFO("%d%s", device_ids[i], (i < device_ids.size() - 1) ? "," : "");
    }
    GGML_LOG_INFO("\n");
    return true;
}

void ggml_backend_cuda_tp_context::cleanup() {
    if (nccl_initialized) {
        ggml_cuda_nccl_cleanup();
        nccl_initialized = false;
    }
}



bool ggml_cuda_tp_available() {
    return g_cuda_tp_ctx != nullptr && g_cuda_tp_ctx->config.enabled;
}

const ggml_tp_config& ggml_cuda_tp_get_config() {
    static ggml_tp_config default_config;
    if (g_cuda_tp_ctx) {
        return g_cuda_tp_ctx->config;
    }
    return default_config;
}

// C interface functions for external linkage
extern "C" {
bool ggml_cuda_tp_init(int tp_size, const int* device_ids, int num_devices) {
    if (tp_size <= 1) {
        return true;
    }

    std::vector<int> device_vec(device_ids, device_ids + num_devices);
    g_cuda_tp_ctx = std::make_unique<ggml_backend_cuda_tp_context>(tp_size, device_vec);
    return g_cuda_tp_ctx->init();
}

void ggml_cuda_tp_cleanup() {
    g_cuda_tp_ctx.reset();
}
}
